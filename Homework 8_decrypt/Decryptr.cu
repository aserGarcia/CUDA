#include "hip/hip_runtime.h"
//nvcc GarciaAserHW8.cu -o temp -lglut -lm -lGLU -lGL

/*------------------------------------
		TO DO
1. dev_r to float3 to increase speed
2. Global vars -> #define
3. Redo lines 118 to 131
------------------------------------*/

#include <GL/glut.h>
#include <stdio.h>
#include <stdlib.h>
#define CHUNKSIZE   1024

int BLOCKSIZE = 265;
int WINDOW_SIZE = 1024;
int FULL_DATA_SIZE = WINDOW_SIZE*WINDOW_SIZE*3; //each pixel has three floats

/*-------------------------------------------
		KERNEL
-------------------------------------------*/
__global__ void kernel(float *a, float *b, float *r){

    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    int offset = idx*3;
//Checking pixel difference between files
    if(idx < CHUNKSIZE){
        if (a[offset+0] != b[offset+0] || a[offset+1] != b[offset+1] || a[offset+2] != b[offset+2]){
            r[offset+0] = 1.0;
            r[offset+1] = 1.0;
            r[offset+2] = 1.0;
        }
        else{
            r[offset+0] = 0.0;
            r[offset+1] = 0.0;
            r[offset+2] = 0.0;
        }
    }
}

/*-------------------------------------------
	DISPLAY TO SCREEN
-------------------------------------------*/
void display()
{
//*********   READING FILE   ***********
    float *file1, *file2;
    FILE *bitmapFile;
    
    hipHostAlloc(&file1, FULL_DATA_SIZE*sizeof(float), hipHostMallocDefault);
    hipHostAlloc(&file2, FULL_DATA_SIZE*sizeof(float), hipHostMallocDefault);

    bitmapFile = fopen("File1", "rb");
    fread(file1, sizeof(float), FULL_DATA_SIZE, bitmapFile);
    bitmapFile = fopen("File2", "rb");
    fread(file2, sizeof(float), FULL_DATA_SIZE, bitmapFile);

    fclose(bitmapFile);

//*********   KERNEL CALL   ***********
    hipStream_t stream0, stream1;
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);

    float *result;
    float *dev_a0, *dev_b0, *dev_r0; //GPU stream 0
    float *dev_a1, *dev_b1, *dev_r1; //GPU stream 1

    hipMalloc(&dev_a0, CHUNKSIZE*3*sizeof(float));
    hipMalloc(&dev_b0, CHUNKSIZE*3*sizeof(float));
    hipMalloc(&dev_r0, CHUNKSIZE*3*sizeof(float));
    hipMalloc(&dev_a1, CHUNKSIZE*3*sizeof(float));
    hipMalloc(&dev_b1, CHUNKSIZE*3*sizeof(float));
    hipMalloc(&dev_r1, CHUNKSIZE*3*sizeof(float));

    hipHostAlloc(&result, FULL_DATA_SIZE*sizeof(float), hipHostMallocDefault);

    for(long i =0; i<CHUNKSIZE*(CHUNKSIZE*3-2); i += CHUNKSIZE*2){
        //copying up to stream0
        hipMemcpyAsync(dev_a0, file1+i,
                        CHUNKSIZE*3*sizeof(float),
                        hipMemcpyHostToDevice,
                        stream0);
        hipMemcpyAsync(dev_a1, file1+i+CHUNKSIZE,
                        CHUNKSIZE*3*sizeof(float),
                        hipMemcpyHostToDevice,
                        stream1);

        //copying up to stream1
        hipMemcpyAsync(dev_b0, file2+i,
                        CHUNKSIZE*3*sizeof(float),
                        hipMemcpyHostToDevice,
                        stream0);
        hipMemcpyAsync(dev_b1, file2+i+CHUNKSIZE,
                        CHUNKSIZE*3*sizeof(float),
                        hipMemcpyHostToDevice,
                        stream1);

        kernel<<<CHUNKSIZE/256, 256, 0, stream0>>>(dev_a0, dev_b0, dev_r0);
        kernel<<<CHUNKSIZE/256, 256, 0, stream1>>>(dev_a1, dev_b1, dev_r1);

        hipMemcpyAsync(result+i, dev_r0,
                        CHUNKSIZE*3*sizeof(float),
                        hipMemcpyDeviceToHost,
                        stream0);
        hipMemcpyAsync(result+i+CHUNKSIZE, dev_r1,
                        CHUNKSIZE*3*sizeof(float),
                        hipMemcpyDeviceToHost,
                        stream1);
        
    }

    hipStreamSynchronize(stream0);
    hipStreamSynchronize(stream1);


    float tol = 0.001;
    float px;
    for(int i =0; i < CHUNKSIZE*3; i+=3){
        for(int j =0; j<CHUNKSIZE*3; j+=3){
            if(result[i+j*CHUNKSIZE]){
                px = file2[i+j*CHUNKSIZE];
                for(int j = 1; j<27; j++){
                    if (abs(px-1.0/float(j))<tol){
                        printf("%c", (char)(j+64));
                    }
                    
                }
                printf(" ");
            }
        }
    }

	glDrawPixels(WINDOW_SIZE, WINDOW_SIZE, GL_RGB, GL_FLOAT, result);
    glFlush();
    
    hipHostFree(file1);
    hipHostFree(file2);
    hipHostFree(result);
    hipFree(dev_a0);
    hipFree(dev_b0);
    hipFree(dev_r0);
    hipFree(dev_a1);
    hipFree(dev_b1);
    hipFree(dev_r1);
    hipStreamDestroy(stream0);
    hipStreamDestroy(stream1);
}

int main(int argc, char** argv)
{
   	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE);
   	glutInitWindowSize(WINDOW_SIZE, WINDOW_SIZE);
   	glutCreateWindow("BitMap");
   	glutDisplayFunc(display);
   	glutMainLoop();
}
